#include "hip/hip_runtime.h"
#include<stdio.h>
#include"scrImagePgmPpmPackage.h"

class GpuTimer
{
      hipEvent_t start;
      hipEvent_t stop;

      public:
 
      GpuTimer()
      {
            hipEventCreate(&start);
            hipEventCreate(&stop);
      }
 
      ~GpuTimer()
      {
            hipEventDestroy(start);
            hipEventDestroy(stop);
      }
 
      void Start()
      {
            hipEventRecord(start, 0);
      }
 
      void Stop()
      {
            hipEventRecord(stop, 0);
      }
 
      float Elapsed()
      {
            float elapsed;
            hipEventSynchronize(stop);
            hipEventElapsedTime(&elapsed, start, stop);
            return elapsed;
      }
};



//Kernel which calculate the resized image
__global__ void createResizedImage(unsigned char *imageScaledData, int scaled_width, float scale_factor, hipTextureObject_t texObj)
{
	const unsigned int tidX = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int tidY = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned index = tidY * scaled_width + tidX;
       	
	// Step 4: Read the texture memory from your texture reference in CUDA Kernel
	// printf("index: %d\n", index);
	imageScaledData[index] = tex2D<unsigned char>(texObj,(float)(tidX * scale_factor),(float)(tidY * scale_factor));
}

int main(int argc, char*argv[])
{
	int height=0, width =0, scaled_height=0,scaled_width=0;
	//Define the scaling ratio	
	float scaling_ratio;//=2.0;
	unsigned char*data;
	unsigned char*scaled_data,*d_scaled_data;

	char inputStr[1024]; //= {"aerosmith-double.pgm"};
	char outputStr[1024];// = {"aerosmith-double-scaled.pgm"};

	if (argc < 5) {
        fprintf(stderr, "expected: <filename> <scaling_ratio> <blockDim.x> <blockDim.y>\n");
        exit(1);
    }

	strcpy(inputStr, argv[1]);
	strcat(inputStr, ".pgm");

	strcpy(outputStr, argv[1]);
	strcat(outputStr, "-scaled.pgm");

	scaling_ratio = atof(argv[2]);
	int block_x = atoi(argv[3]); //32
	int block_y = atoi(argv[4]); //32
	int grid_x, grid_y;

	hipError_t returnValue;

	//Create a channel Description to be used while linking to the tecture
	hipArray* cu_array;
	hipChannelFormatKind kind = hipChannelFormatKindUnsigned;
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(8, 0, 0, 0, kind);

	get_PgmPpmParams(inputStr, &height, &width);	//getting height and width of the current image
	data = (unsigned char*)malloc(height*width*sizeof(unsigned char));
	// printf("\n Reading image width height and width [%d][%d]", height, width);
	scr_read_pgm( inputStr , data, height, width );//loading an image to "inputimage"

	scaled_height = (int)(height*scaling_ratio);
	scaled_width = (int)(width*scaling_ratio);
	scaled_data = (unsigned char*)malloc(scaled_height*scaled_width*sizeof(unsigned char));
	// printf("\n scaled image width height and width [%d][%d]", scaled_height, scaled_width);

	//Allocate CUDA Array
 	returnValue = hipMallocArray( &cu_array, &channelDesc, width, height);
  if(returnValue != hipSuccess)
  	printf("\n%s", hipGetErrorString(returnValue));
	// hipMemcpyToArray()
	returnValue = (hipError_t)(returnValue | hipMemcpyToArray( cu_array, 0, 0, data, height * width * sizeof(unsigned char), hipMemcpyHostToDevice));
	// printf("\n%s", hipGetErrorString(returnValue));

	if(returnValue != hipSuccess)
		printf("\n Got error while running CUDA API Array Copy");

	// Step 1. Specify texture
	struct hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = cu_array;

	// Step 2. Specify texture object parameters
	struct hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(texDesc));
	texDesc.addressMode[0] = hipAddressModeClamp;
	texDesc.addressMode[1] = hipAddressModeClamp;
	texDesc.filterMode = hipFilterModePoint;
	texDesc.readMode = hipReadModeElementType;
	texDesc.normalizedCoords = 0;

	// Step 3: Create texture object
	hipTextureObject_t texObj = 0;
	hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);

	if(returnValue != hipSuccess) 
		printf("\n Got error while running CUDA API Bind Texture");
	
	hipMalloc(&d_scaled_data, scaled_height*scaled_width*sizeof(unsigned char) );

	dim3 dimBlock(block_x, block_y, 1);

	grid_x = scaled_width / dimBlock.x + 1;
	grid_y = scaled_height / dimBlock.y + 1;
	dim3 dimGrid(grid_x, grid_y, 1);
	// printf("\n Launching grid with blocks [%d][%d] \n", dimGrid.x,dimGrid.y);

	GpuTimer *timer = new GpuTimer();
	timer->Start();
	createResizedImage<<<dimGrid, dimBlock>>>(d_scaled_data, scaled_width, 1.0 / scaling_ratio, texObj);
	returnValue = (hipError_t)(returnValue | hipDeviceSynchronize());

	returnValue = (hipError_t)(returnValue |hipMemcpy (scaled_data , d_scaled_data, scaled_height*scaled_width*sizeof(unsigned char), hipMemcpyDeviceToHost ));
	if(returnValue != hipSuccess) 
		printf("\n Got error while running CUDA API kernel");

	timer->Stop();
	printf("%s,%d,%d,%d,%d,%d,%d,%f,%d,%d,%f\n",argv[1],block_x, block_y, dimGrid.x, dimGrid.y, height, width, scaling_ratio, scaled_height, scaled_width, timer->Elapsed());

	// Step 5: Destroy texture object
	hipDestroyTextureObject(texObj);
	
	scr_write_pgm( outputStr, scaled_data, scaled_height, scaled_width, "####" ); //storing the image with the detections
		
	if(data != NULL)
		free(data);
	if(cu_array !=NULL)
		hipFreeArray(cu_array);
	if(scaled_data != NULL)
		free(scaled_data);
	if(d_scaled_data!=NULL)
		hipFree(d_scaled_data);
	
	delete(timer);
	return 0;
}
